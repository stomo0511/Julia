#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
//#include <cv.hpp>

#include <thrust/complex.h>

#define EPS 0.0000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 1.0      // 初期値の最大絶対値
#define ZOOM 600      // 拡大率
#define RMAX 2000     // 複素平面の分割数

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

#define NFP 4  // 不動点の数
thrust::complex<double> fps[NFP];

void setZero( thrust::complex<double> *fps )
{
	fps[0] = thrust::complex<double> (  1.0,  0.0 );
	fps[1] = thrust::complex<double> (  0.0,  0.5 );
	fps[2] = thrust::complex<double> (  0.0, -0.5 );
	fps[3] = thrust::complex<double> ( -1.0,  0.0 );
}

template<typename T> thrust::complex<T> vf( thrust::complex<T> z )
{
	return z*z*z*z -(3.0*z*z)/(4.0) -(1.0)/(4.0);
}

template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	return 4.0*z*z*z - (3.0*z)/(2.0);
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z -= vf(z) / df(z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int col = 0;
	double min = (double)(MAXIT);

	for (int i=0; i<NFP; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を黒に
	glClearColor(0.0, 0.0, 0.0, 1.0); // 塗りつぶしの色を指定（黒）
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	setZero(fps);      // 零点のセット

//	double x = (double)(-ZMAX);
	double x = (double)0.0;

	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-1.0);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			//double p;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			double brit;
			if (count > 13)
				brit = 0.0;
			else
				brit = (13.0 - double(count)) / 13.0;
			// 明るさの補正
			brit += 0.15;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			case 3:
				glColor3d(brit,0.0,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real()-1.0, z0.imag() );  // 点の描画（原点補正あり）

//			if (count == 4)
//			{
//				std::cout << "z0 = " << z0 << ", count = " << count;
//				std::cout << ", z = " << z << ", bright = " << brit;
//				std::cout << ", color = " << FixPoint(z) << ", p = " << p;
//				std::cout << ", er = " << er << std::endl;
//			}
			y += (double)(2.0 / RMAX);
		}
		x += (double)(2.0 / RMAX);
	}

	glEnd();
	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
//	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

void saveImage( const unsigned int imageWidth, const unsigned int imageHeight )
{
	const unsigned int channelNum = 3; // RGBなら3, RGBAなら4
	void* dataBuffer = ( GLubyte* )malloc( imageWidth * imageHeight * channelNum );

	// 読み取るOpneGLのバッファを指定 GL_FRONT:フロントバッファ　GL_BACK:バックバッファ
	glReadBuffer( GL_FRONT );

	// OpenGLで画面に描画されている内容をバッファに格納
	glReadPixels(
			0,                 //読み取る領域の左下隅のx座標
			0,                 //読み取る領域の左下隅のy座標 //0 or getCurrentWidth() - 1
			imageWidth,             //読み取る領域の幅
			imageHeight,            //読み取る領域の高さ
			GL_RGB, //it means GL_BGR,           //取得したい色情報の形式
			GL_UNSIGNED_BYTE,  //読み取ったデータを保存する配列の型
			dataBuffer      //ビットマップのピクセルデータ（実際にはバイト配列）へのポインタ
	);
	glFlush();

//	GLubyte* p = static_cast<GLubyte*>( dataBuffer );
//	std::string fname = "outputImage.jpg";
//	IplImage* outImage = cvCreateImage( cvSize( imageWidth, imageHeight ), IPL_DEPTH_8U, 3 );
//
//	for ( unsigned int j = 0; j < imageHeight; ++ j )
//	{
//		for ( unsigned int i = 0; i < imageWidth; ++i )
//		{
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 0 ] = *p;
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 1 ] = *( p + 1 );
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 2 ] = *( p + 2 );
//			p += 3;
//		}
//	}
//
//	cvSaveImage( fname.c_str(), outImage );
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
