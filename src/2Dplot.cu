#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 16      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 1000     // 複素平面の分割数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  0.0,  1.0 ),
	thrust::complex<double> (  1.0,  2.0 ),
	thrust::complex<double> ( -1.0,  2.0 ),
	thrust::complex<double> (  3.0, -3.0 ),
	thrust::complex<double> ( -3.0, -3.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0,   0.0 ),  // z^5
	thrust::complex<double> (  0.0,   1.0 ),  // Z^4
	thrust::complex<double> (  3.0,   0.0 ),  // Z^3
	thrust::complex<double> (  0.0,  41.0 ),  // z^2
	thrust::complex<double> (132.0,   0.0 ),  // z^1
	thrust::complex<double> (  0.0, -90.0 )   // z^0
};

// Hornet method for polynomial
template<typename T> void Horner( std::vector< thrust::complex<T> > cf, thrust::complex<T> z,
					thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = Cef[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = Cef.begin()+1; itr < Cef.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	thrust::complex<T> vf, df;
	Horner( Cef, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z -= vf / df;
		Horner( Cef, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)MAXIT;

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs(z - *itr) < min)
		{
			min = abs(z - *itr);
			col = i;
		}
		i++;
	}

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			double bright;
			if (count > MAXIT)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(MAXIT - (count-1)) / double(MAXIT);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}
			glVertex2d( z0.real(), z0.imag() );  // 点の描画

			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( (*itr).real(), (*itr).imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	glFlush();
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
