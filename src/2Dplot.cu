#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>

#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 40      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 2000     // 複素平面の分割数

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

template<typename T> thrust::complex<T> vf( thrust::complex<T> z )
{
	thrust::complex<T> iu = thrust::complex<T> ( 0.0, 1.0 );
	return z*z*z*z*z + iu*z*z*z*z + 3.0*z*z*z + 41.0*iu*z*z + 132.0*z -90.0*iu;
}

template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	thrust::complex<T> iu = thrust::complex<T> ( 0.0, 1.0 );
	return 5.0*z*z*z*z + 4.0*iu*z*z*z + 9.0*z*z + 82.0*iu*z + 132.0;
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count )
{
	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z -= vf(z) / df(z);
		count++;
	}

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	const int nfp = 5;  // 不動点の数
	thrust::complex<T> *fps = new thrust::complex<T> [nfp];

	fps[0] = thrust::complex<T> (  0.0,  1.0 );
	fps[1] = thrust::complex<T> (  1.0,  2.0 );
	fps[2] = thrust::complex<T> ( -1.0,  2.0 );
	fps[3] = thrust::complex<T> (  3.0, -3.0 );
	fps[4] = thrust::complex<T> ( -3.0, -3.0 );

	int col = 0;
	double min = 0.001;

	for (int i=0; i<nfp; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}
	delete[] fps;

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);

	for (int i=0; i<RMAX; i++)
	{
		int count;
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count);
//			double brit = (double)(1.0/MAXIT)*(MAXIT - count);

			double brit;
			if (count > 13)
				brit = 0.0;
			else
				brit = (13.0 - double(count)) / 13.0;
			// 明るさの補正
			brit += 0.1;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			case 3:
				glColor3d(brit,0.0,brit);
				break;
			case 4:
				glColor3d(0.0,brit,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
//			if (count >= MAXIT-1)
//			{
//				std::cout << "z0 = " << z0 << ", count = " << count;
//				std::cout << ", z = " << z << ", bright = " << brit;
//				std::cout << ", color = " << FixPoint(z) << std::endl;
//			}
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
