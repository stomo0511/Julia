#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>

#include <thrust/complex.h>

#define EPS 0.0000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 0.8      // 初期値の最大絶対値
#define ZOOM 600      // 拡大率
#define RMAX 2000     // 複素平面の分割数
#define XOFS 1.0      // x軸のオフセット

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

#define NFP 4  // 不動点の数
thrust::complex<double> fps[NFP];

void setZero( thrust::complex<double> *fps )
{
	fps[0] = thrust::complex<double> (  1.0,  0.0 );
	fps[1] = thrust::complex<double> (  0.0,  0.5 );
	fps[2] = thrust::complex<double> (  0.0, -0.5 );
	fps[3] = thrust::complex<double> ( -1.0,  0.0 );
}

template<typename T> thrust::complex<T> vf( thrust::complex<T> z )
{
	return z*z*z*z -(3.0*z*z)/(4.0) -(1.0)/(4.0);
}

template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	return 4.0*z*z*z - (3.0*z)/(2.0);
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z -= vf(z) / df(z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int col = 0;
	double min = (double)(MAXIT);

	for (int i=0; i<NFP; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}

	return col;
}

void DrawApollonius( int i, int j, double alp )
{
	const int pts = 180;    // 円周上の点数

	thrust::complex<double> center = (fps[i] - alp*alp*fps[j]) / (1.0 - alp*alp);
	double radius = alp*abs(fps[i] - fps[j]) / (1.0 - alp*alp);
	double tic = (double)(2.0*M_PI / pts);

	// Z_i の描画
	glColor3d(1.0,1.0,1.0);   // 白の点を描画
	glPointSize(10.0);      // 点の大きさ（ディフォルトは1.0)
	glBegin(GL_POINTS);
	glVertex2d( fps[i].real() -XOFS, fps[i].imag() );
	glEnd();

	// Apollonius円の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(2.0);         // 線の太さ（ディフォルトは1.0）

	glBegin(GL_LINE_LOOP);
	for (int i=1; i<pts; i++)
	{
		glVertex2d( center.real() + radius*cos( tic*i ) -XOFS  , center.imag() + radius*sin( tic*i ) );
	}
	glEnd();
	glFlush();
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を黒に
	glClearColor(0.0, 0.0, 0.0, 1.0); // 塗りつぶしの色を指定（黒）
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	setZero(fps);      // 零点のセット

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

//	double x = (double)(-ZMAX);
//	double x = (double)0.0;
	double x = (double)(-ZMAX + XOFS);

	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			//double p;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			double brit;
			if (count > 13)
				brit = 0.0;
			else
				brit = (13.0 - double(count)) / 13.0;
			// 明るさの補正
			brit += 0.15;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			case 3:
				glColor3d(brit,0.0,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real()-XOFS, z0.imag() );  // 点の描画（原点補正あり）

			y += (double)(2.0*ZMAX / RMAX);
		}
		x += (double)(2.0*ZMAX / RMAX);
	}

	glEnd();
	glFlush();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apollonius円の描画
	double alp = (double)1.0 / (2*NFP - 3.0);

	for (int j=1; j<NFP; j++)
	{
		DrawApollonius( 0, j, alp );
	}
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

void saveImage( const unsigned int imageWidth, const unsigned int imageHeight )
{
	const unsigned int channelNum = 3; // RGBなら3, RGBAなら4
	void* dataBuffer = ( GLubyte* )malloc( imageWidth * imageHeight * channelNum );

	// 読み取るOpneGLのバッファを指定 GL_FRONT:フロントバッファ　GL_BACK:バックバッファ
	glReadBuffer( GL_FRONT );

	// OpenGLで画面に描画されている内容をバッファに格納
	glReadPixels(
			0,                 //読み取る領域の左下隅のx座標
			0,                 //読み取る領域の左下隅のy座標 //0 or getCurrentWidth() - 1
			imageWidth,             //読み取る領域の幅
			imageHeight,            //読み取る領域の高さ
			GL_RGB, //it means GL_BGR,           //取得したい色情報の形式
			GL_UNSIGNED_BYTE,  //読み取ったデータを保存する配列の型
			dataBuffer      //ビットマップのピクセルデータ（実際にはバイト配列）へのポインタ
	);
	glFlush();

//	GLubyte* p = static_cast<GLubyte*>( dataBuffer );
//	std::string fname = "outputImage.jpg";
//	IplImage* outImage = cvCreateImage( cvSize( imageWidth, imageHeight ), IPL_DEPTH_8U, 3 );
//
//	for ( unsigned int j = 0; j < imageHeight; ++ j )
//	{
//		for ( unsigned int i = 0; i < imageWidth; ++i )
//		{
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 0 ] = *p;
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 1 ] = *( p + 1 );
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 2 ] = *( p + 2 );
//			p += 3;
//		}
//	}
//
//	cvSaveImage( fname.c_str(), outImage );
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
