#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <cstdlib>
#include <cmath>
#include <unistd.h>
#include <vector>

#include <thrust/complex.h>

#define EPS 0.0000001  // 停止判定
#define MAXIT 40    // 最大反復回数
#define ZMAX 1.5     // 初期値の最大絶対値
#define ZOOM 500     // 拡大率
#define RMAX 4000    // 複素平面の分割数
#define OFFS 0.3     // 明度のオフセット

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

template<typename T> thrust::complex<T> Update( thrust::complex<T> z )
{
	thrust::complex<T> vf = z*z*z -1.0;
	thrust::complex<T> df = 3.0*z*z;

	return z - vf / df;
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count )
{
	double diff = (double)(MAXIT);

	count = 0;
	while ((count < MAXIT) && (diff > EPS))
	{
		thrust::complex<T> d = Update(z);
		diff = abs( d - z );
		count++;
		z = d;
//		std::cout << z << std::endl;
	}

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	const int nfp = 3;  // 不動点の数
	thrust::complex<T> *fps = new thrust::complex<T> [nfp];

	fps[0] = thrust::complex<T> ( 1.0, 0.0 );
	fps[1] = thrust::complex<T> ( -0.5, 0.866025 );
	fps[2] = thrust::complex<T> (  0.5, 0.866025 );

	int col = 0;
	double min = (double)(MAXIT);

	for (int i=0; i<nfp; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[0]);
			col = i;
		}
	}
	delete[] fps;

	return col;
}

void display(void)
{
	glClearColor(0.0, 0.0, 0.0, 1.0); // 塗りつぶしの色を指定（黒）
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	///////////////////////////////////
	// 座標軸の描画
	glBegin(GL_LINES);
	glColor3d(1.0, 0.0, 0.0);         // 赤 (1,0,0) で描画
	glVertex2d(-ZMAX,  0.0);
	glVertex2d( ZMAX,  0.0);

	glVertex2d( 0.0, -ZMAX);
	glVertex2d( 0.0,  ZMAX);

	glEnd();
	glFlush();                        // OpenGL命令のフラッシュ
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);

//	#pragma omp parallel for
	for (int i=0; i<RMAX; i++)
	{
		int count;
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count);
			double brit = (double)(1.0/MAXIT)*(MAXIT - count);
			//double brit = (1.0/MAXIT)*count + OFFS;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			if (count >= MAXIT-10)
			{
				std::cout << "z0 = " << z0 << ", count = " << count;
				std::cout << ", z = " << z << ", bright = " << brit;
				std::cout << ", color = " << FixPoint(z) << std::endl;
			}
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitDisplayMode(GLUT_RGBA); // RGBモードに設定
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
