#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>

#include <thrust/complex.h>

#define EPS 0.0000001  // 停止判定
#define MAXIT 40      // 最大反復回数
#define ZMAX 1.5      // 初期値の最大絶対値
#define ZOOM 500      // 拡大率
#define RMAX 2000     // 複素平面の分割数

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

template<typename T> thrust::complex<T> vf( thrust::complex<T> z )
{
	return z*z*z -1.0;
}

template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	return 3.0*z*z;
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z -= vf(z) / df(z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	const int nfp = 3;  // 不動点の数
	thrust::complex<T> *fps = new thrust::complex<T> [nfp];

	fps[0] = thrust::complex<T> ( 1.0, 0.0 );
	fps[1] = thrust::complex<T> ( -0.5,  0.866025 );
	fps[2] = thrust::complex<T> ( -0.5, -0.866025 );

	int col = 0;
	double min = (double)(MAXIT);

	for (int i=0; i<nfp; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}
	delete[] fps;

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);

	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er, p;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			//			double brit = (double)(1.0/MAXIT)*(MAXIT - count);

//			p = 0.0;
//			if (count < MAXIT)
//			{
//				p = log2( -12.0 / log10(er));
//			}
//			double brit = p;

			double brit;
			if (count > 13)
				brit = 0.0;
			else
				brit = (13.0 - double(count)) / 13.0;
			// 明るさの補正
			brit += 0.1;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			//if (count >= MAXIT-4)
//			if (count == 4)
//			{
//				std::cout << "z0 = " << z0 << ", count = " << count;
//				std::cout << ", z = " << z << ", bright = " << brit;
//				std::cout << ", color = " << FixPoint(z) << ", p = " << p;
//				std::cout << ", er = " << er << std::endl;
//			}
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
