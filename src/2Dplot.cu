#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.0000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 0.8      // 初期値の最大絶対値
#define ZOOM 600      // 拡大率
#define RMAX 1500     // 複素平面の分割数
#define XOFS 1.0      // x軸のオフセット

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  1.0,  0.0 ),
	thrust::complex<double> (  0.0,  0.5 ),
	thrust::complex<double> (  0.0, -0.5 ),
	thrust::complex<double> ( -1.0,  0.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (      1.0, 0.0 ),  // Z^4
	thrust::complex<double> (      0.0, 0.0 ),  // Z^3
	thrust::complex<double> ( -3.0/4.0, 0.0 ),  // z^2
	thrust::complex<double> (      0.0, 0.0 ),  // z^1
	thrust::complex<double> ( -1.0/4.0, 0.0 )   // z^0
};

template<typename T> thrust::complex<T> vf( thrust::complex<T> cf, thrust::complex<T> z )
{
	thrust::omplex<T> tmp = cf[0];
    for(auto itr = cf.begin()+1; itr != cf.end(); ++itr)
    {
    	tmp = tmp*z + *itr;
    }
	return tmp;
}

template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	return 4.0*z*z*z - (3.0*z)/(2.0);
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z -= vf(z) / df(z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int col = 0;
	double min = (double)(MAXIT);

	for (int i=0; i<NFP; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}

	return col;
}

void DrawApollonius( int i, int j, double alp )
{
	const int pts = 180;    // 円周上の点数

	thrust::complex<double> center = (fps[i] - alp*alp*fps[j]) / (1.0 - alp*alp);
	double radius = alp*abs(fps[i] - fps[j]) / (1.0 - alp*alp);
	double tic = (double)(2.0*M_PI / pts);

	// Z_i の描画
	glColor3d(1.0,1.0,1.0);   // 白の点を描画
	glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
	glBegin(GL_POINTS);
	glVertex2d( fps[i].real() -XOFS, fps[i].imag() );
	glEnd();

	// Apollonius円の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(2.0);         // 線の太さ（ディフォルトは1.0）

	glBegin(GL_LINE_LOOP);
	for (int i=1; i<pts; i++)
	{
		glVertex2d( center.real() + radius*cos( tic*i ) -XOFS  , center.imag() + radius*sin( tic*i ) );
	}
	glEnd();
	glFlush();
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定（黒）
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	setZero(fps);      // 零点のセット

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX + XOFS);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			//double p;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real()-XOFS, z0.imag() );  // 点の描画（原点補正あり）

			y += (double)(2.0*ZMAX / RMAX);
		}
		x += (double)(2.0*ZMAX / RMAX);
	}

	glEnd();
	glFlush();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apollonius円の描画
	double alp = (double)1.0 / (2*NFP - 3.0);

	for (int j=1; j<NFP; j++)
	{
		DrawApollonius( 0, j, alp );
	}
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

void saveImage( const unsigned int imageWidth, const unsigned int imageHeight )
{
	const unsigned int channelNum = 3; // RGBなら3, RGBAなら4
	void* dataBuffer = ( GLubyte* )malloc( imageWidth * imageHeight * channelNum );

	// 読み取るOpneGLのバッファを指定 GL_FRONT:フロントバッファ　GL_BACK:バックバッファ
	glReadBuffer( GL_FRONT );

	// OpenGLで画面に描画されている内容をバッファに格納
	glReadPixels(
			0,                 //読み取る領域の左下隅のx座標
			0,                 //読み取る領域の左下隅のy座標 //0 or getCurrentWidth() - 1
			imageWidth,             //読み取る領域の幅
			imageHeight,            //読み取る領域の高さ
			GL_RGB, //it means GL_BGR,           //取得したい色情報の形式
			GL_UNSIGNED_BYTE,  //読み取ったデータを保存する配列の型
			dataBuffer      //ビットマップのピクセルデータ（実際にはバイト配列）へのポインタ
	);
	glFlush();

//	GLubyte* p = static_cast<GLubyte*>( dataBuffer );
//	std::string fname = "outputImage.jpg";
//	IplImage* outImage = cvCreateImage( cvSize( imageWidth, imageHeight ), IPL_DEPTH_8U, 3 );
//
//	for ( unsigned int j = 0; j < imageHeight; ++ j )
//	{
//		for ( unsigned int i = 0; i < imageWidth; ++i )
//		{
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 0 ] = *p;
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 1 ] = *( p + 1 );
//			outImage->imageData[ ( imageHeight - j - 1 ) * outImage->widthStep + i * 3 + 2 ] = *( p + 2 );
//			p += 3;
//		}
//	}
//
//	cvSaveImage( fname.c_str(), outImage );
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(500,500);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
