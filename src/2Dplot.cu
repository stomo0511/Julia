#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 2.0      // 初期値の最大絶対値
#define ZOOM 300      // 拡大率
#define RMAX 1000     // 複素平面の分割数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  1.0,  0.0 ),
	thrust::complex<double> ( -0.5,  0.866025 ),
	thrust::complex<double> ( -0.5, -0.866025 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0, 0.0 ),  // Z^3
	thrust::complex<double> (  0.0, 0.0 ),  // z^2
	thrust::complex<double> (  0.0, 0.0 ),  // z^1
	thrust::complex<double> ( -1.0, 0.0 )   // z^0
};

// Hornet method for polynomial
template<typename T> void Horner( std::vector< thrust::complex<T> > cf, thrust::complex<T> z,
					thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = Cef[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = Cef.begin()+1; itr < Cef.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

template<typename T> thrust::complex<T> Newton( thrust::complex<T> z, int &count, double &er )
{
	thrust::complex<T> vf, df;
	Horner( Cef, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z -= vf / df;
		Horner( Cef, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)(MAXIT);

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs( z - *itr) < min)
		{
			min = abs( z - *itr);
			col = i;
		}
		i++;
	}
	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);

	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Newton(z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();

	//////////////////////////////////////////////
	// 零点の描画
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( (*itr).real(), (*itr).imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -w/ZOOM, w/ZOOM, -h/ZOOM, h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
