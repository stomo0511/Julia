#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 1000     // 複素平面の分割数

int P;  // Nourein法の次数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  0.0,  1.0 ),
	thrust::complex<double> (  1.0,  2.0 ),
	thrust::complex<double> ( -1.0,  2.0 ),
	thrust::complex<double> (  3.0, -3.0 ),
	thrust::complex<double> ( -3.0, -3.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0,   0.0 ),  // z^5
	thrust::complex<double> (  0.0,   1.0 ),  // Z^4
	thrust::complex<double> (  3.0,   0.0 ),  // Z^3
	thrust::complex<double> (  0.0,  41.0 ),  // z^2
	thrust::complex<double> (132.0,   0.0 ),  // z^1
	thrust::complex<double> (  0.0, -90.0 )   // z^0
};

std::vector<double> Gam( Zrs.size() );  // Γ
std::vector<double> Alp( Zrs.size() );  // α

// Hornet method for polynomial
template<typename T> void Horner( std::vector< thrust::complex<T> > cf, thrust::complex<T> z,
					thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = Cef[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = Cef.begin()+1; itr < Cef.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

// Nourein subfunction
template <typename T> thrust::complex<T> vc( const int K, thrust::complex<T> z )
{
	thrust::complex<T> tmp = thrust::complex<T> (0.0,0.0);;

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		thrust::complex<T> vf, df;
		Horner( Cef, *itr, vf, df );

		// tmp *= (1/f'(z_i) (-1 / (z_i -z)^{K+1})
		tmp += ( (T)(1.0) / df )*( (T)(-1.0) / pow( (*itr - z), (T)(K+1) ));
	}
	return tmp;
}

template <typename T> thrust::complex<T> Nourein( const int p, thrust::complex<T> z, int &count, T &er )
{
	assert(p>=2);

	thrust::complex<T> vf, df;
	Horner( Cef, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z += vc(p-2,z) / vc(p-1,z);
		Horner( Cef, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

template <typename T> void SetGamma( std::vector<T> &Gam )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T max = (T)(0.0);
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i != j)
			{
				thrust::complex<T> vf, dfi, dfj;

				Horner( Cef, Zrs[i], vf, dfi );
				Horner( Cef, Zrs[j], vf, dfj );

				if (max < abs(dfi / dfj))
				{
					max = abs(dfi / dfj);
				}
			}
		}
		Gam[i] = max;
	}
}

template <typename T> T fAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	return ((T)(Zrs.size()) -1.0) * Gamma * pow(alp,(T)(p-1)) - (1.0-alp)/(1.0+3.0*alp);
}

template <typename T> T dAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	T tmp = ((T)(Zrs.size()) -1.0) * Gamma * ((T)(p) -1.0) * pow(alp,(T)(p-2));
	return  tmp + 4.0/(1.0+6.0*alp+9.0*alp*alp);
}

template <typename T> void GetAlpha( const std::vector<T> Gam, std::vector<T> &Alp )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T alp = (T)(1.0);
		int count = 0;

		while ((count < MAXIT) && (abs(fAlp(P,Gam[i],alp)) > EPS))
		{
			alp -= fAlp(P,Gam[i],alp) / dAlp(P,Gam[i],alp);
			count++;
		}
		if (count == MAXIT)
		{
			std::cerr << "No convergence in GetAlpha\n";
			std::exit (EXIT_FAILURE);
		}
		Alp[i] = alp;
	}
}

template <typename T> void DrawApollonius( const int i, const int j, const T alp )
{
	const int pts = 180;    // 円周上の点数

	thrust::complex<T> center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
	double radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
	double tic = (double)(2.0*M_PI / pts);

	// Apollonius円の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(1.0);         // 線の太さ（ディフォルトは1.0）

	glBegin(GL_LINE_LOOP);
	for (int i=1; i<pts; i++)
	{
		glVertex2d( center.real() + radius*cos( tic*i )  , center.imag() + radius*sin( tic*i ) );
	}
	glEnd();
	glFlush();
}

template <typename T> void DrawApRegion( const T alp )
{
	// Apollonius領域の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(2.0);         // 線の太さ（ディフォルトは1.0

	const int pts = 120;    // 円周上の点数
	thrust::complex<T> center;
	T radius;
	T start, end, tic;

	int i, j;

	// p=2
	{
		// z_0, z_1
		i = 0; j=1;
		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
		start = -1.41075;
		tic = -2.0*start / pts;

		glBegin(GL_LINE_STRIP);
		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
		for (int t=1; t<=pts; t++)
		{
			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
		}
		glEnd();

		// z_0, z_2
		i = 0; j=2;
		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
		start = 1.73084;
		tic = 2.0*(M_PI - start) / pts;

		glBegin(GL_LINE_STRIP);
		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
		for (int t=1; t<=pts; t++)
		{
			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
		}
		glEnd();
	}

	// p=4
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.22945;
//		tic = -2.0*start / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 1.91214;
//		tic = 2.0*(M_PI - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=8
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.08417;
//		tic = -2.0*start / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 2.05743;
//		tic = 2.0*(M_PI - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=16
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -0.0309316;
//		end   = 0.976624;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start =  2.16497;
//		end   =  2*M_PI - 3.11066;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_3
//		i = 0; j=3;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.0758;
//		end   = -0.928056;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_4
//		i = 0; j=4;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -2.21354;
//		end   = -2.06579;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=32
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 0.325413;
//		end   = 0.902587;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start =  2.23901;
//		end   =  2.81618;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_3
//		i = 0; j=3;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -0.903981;
//		end   = -1.01722;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_4
//		i = 0; j=4;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -2.12437;
//		end   = -2.23761;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}
	glFlush();
}

template <typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)(MAXIT);

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs( z - *itr) < min)
		{
			min = abs( z - *itr);
			col = i;
		}
		i++;
	}
	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(P,z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( (*itr).real(), (*itr).imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apolloniusの描画
//	SetGamma( Gam );       // Γ
//	GetAlpha( Gam, Alp );  // α
//	for (int i=0; i<Zrs.size(); i++)
//	{
//		for (int j=0; j<Zrs.size(); j++)
//		{
//			if (i!=j)
//				DrawApollonius(i,j,Alp[i]);
//		}
//	}
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apollonius領域の描画
	SetGamma( Gam );       // Γ
	GetAlpha( Gam, Alp );  // α

	DrawApRegion( Alp[0] );
	//////////////////////////////////////////////

	glFlush();
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	if (argc<2)
	{
		std::cerr << "Usage: a.out [Order]\n";
		exit (EXIT_FAILURE);
	}
	P = atoi(argv[1]);  // Nourein法の次数

	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
