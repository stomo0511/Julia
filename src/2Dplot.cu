#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>

#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 2000     // 複素平面の分割数
#define ORD  32        // 次数

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

#define NFP 5 // 零点の数
thrust::complex<double> fps[NFP];

void setZero( thrust::complex<double> *fps )
{
	fps[0] = thrust::complex<double> (  0.0,  1.0 );
	fps[1] = thrust::complex<double> (  1.0,  2.0 );
	fps[2] = thrust::complex<double> ( -1.0,  2.0 );
	fps[3] = thrust::complex<double> (  3.0, -3.0 );
	fps[4] = thrust::complex<double> ( -3.0, -3.0 );
}

// Polynomial
template<typename T> thrust::complex<T> vf( thrust::complex<T> z )
{
	thrust::complex<T> iu = thrust::complex<T> ( 0.0, 1.0 );
	return z*z*z*z*z + iu*z*z*z*z + + 3.0*z*z*z + 41.0*iu*z*z + 132.0*z -90.0*iu;
}

// derived function of the polynomial
template<typename T> thrust::complex<T> df( thrust::complex<T> z )
{
	thrust::complex<T> iu = thrust::complex<T> ( 0.0, 1.0 );
	return 5.0*z*z*z*z + 4.0*iu*z*z*z + 9.0*z*z + 82.0*iu*z + 132.0;
}

// Nourein subfunction
template<typename T> thrust::complex<T> vc( const int K, thrust::complex<T> z )
{
	thrust::complex<T> f = thrust::complex<T> (0.0,0.0);;

	for (int i=0; i<NFP; i++)
	{
		thrust::complex<T> tmp = thrust::complex<T> (1.0,0.0);

		// tmp = (z_i -z)^{k+1}
		for (int k=0; k<=K; k++)
		{
			tmp = tmp * (fps[i] - z);
		}
		// tmp = -1.0 /  (z_i -z)^{k+1}
		tmp = -1.0 / tmp;

		f += ( 1.0 / df(fps[i]) )*tmp;
	}
	return f;
}

template<typename T> thrust::complex<T> Nourein( const int p, thrust::complex<T> z, int &count, double &er )
{
	assert(p>=2);

	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z += vc(p-2,z) / vc(p-1,z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

template<typename T> int FixPoint( thrust::complex<T> z )
{
	int col = 0;
//	int col = 1;
	double min = (double)MAXIT;

	for (int i=0; i<NFP; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	setZero(fps);     // 零点のセット

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(ORD,z0,count,er);

			double brit;
			if (count > 13)
				brit = 0.0;
			else
				brit = (13.0 - double(count)) / 13.0;
			// 明るさの補正
			brit += 0.2;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(brit,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,brit,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,brit);
				break;
			case 3:
				glColor3d(brit,0.0,brit);
				break;
			case 4:
				glColor3d(0.0,brit,brit);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
//			if (count >= MAXIT-1)
//			{
//				std::cout << "z0 = " << z0 << ", count = " << count;
//				std::cout << ", z = " << z << ", bright = " << brit;
//				std::cout << ", color = " << FixPoint(z) << std::endl;
//			}
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	glFlush();
	//////////////////////////////////////////////
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
