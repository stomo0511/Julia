#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>

#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 16      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 1000     // 複素平面の分割数

int P;  // Nourein法の次数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  0.0,  1.0 ),
	thrust::complex<double> (  1.0,  2.0 ),
	thrust::complex<double> ( -1.0,  2.0 ),
	thrust::complex<double> (  3.0, -3.0 ),
	thrust::complex<double> ( -3.0, -3.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0,   0.0 ),  // z^5
	thrust::complex<double> (  0.0,   1.0 ),  // Z^4
	thrust::complex<double> (  3.0,   0.0 ),  // Z^3
	thrust::complex<double> (  0.0,  41.0 ),  // z^2
	thrust::complex<double> (132.0,   0.0 ),  // z^1
	thrust::complex<double> (  0.0, -90.0 )   // z^0
};

std::vector<double> Gam( Zrs.size() );  // Γ
std::vector<double> Alp( Zrs.size() );  // α

// Hornet method for polynomial
template<typename T> void Horner(
		const std::vector< thrust::complex<T> > cf,
		const thrust::complex<T> z,
		thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = cf[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = cf.begin()+1; itr < cf.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

// Nourein subfunction
template <typename T> thrust::complex<T> Chi_m(
		const std::vector< thrust::complex<T> > zr,
		const std::vector< thrust::complex<T> > cf,
		const int m,
		const thrust::complex<T> z )
{
	thrust::complex<T> tmp = thrust::complex<T> (0.0,0.0);

	for (auto itr = zr.begin(); itr < zr.end(); ++itr )
	{
		thrust::complex<T> vf, df;
		Horner( cf, *itr, vf, df );

		// tmp *= (1/f'(z_i) (-1 / (z_i -z)^{m+1})
		tmp += ( (T)(1.0) / df )*( (T)(-1.0) / pow( (*itr - z), (T)(m+1) ));
	}
	return tmp;
}

template <typename T> thrust::complex<T> Nourein(
		const std::vector< thrust::complex<T> > zr,
		const std::vector< thrust::complex<T> > cf,
		const int p,
		thrust::complex<T> z,
		int &count, T &er )
{
	assert(p>=2);

	thrust::complex<T> vf, df;
	Horner( cf, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z += Chi_m(zr,cf,p-2,z) / Chi_m(zr,cf,p-1,z);
		Horner( cf, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

// 円の描画
template <typename T> void Circle2D(T r,int x,int y)
{
	for (T th1 = 0.0;  th1 <= 360.0;  th1 = th1 + 1.0)
	{
		T th2 = th1 + 10.0;
		T th1_rad = th1 / 180.0 * M_PI;
		T th2_rad = th2 / 180.0 * M_PI;

		T x1 = r * cos(th1_rad);
		T y1 = r * sin(th1_rad);
		T x2 = r * cos(th2_rad);
		T y2 = r * sin(th2_rad);

		glBegin(GL_LINES);
		glVertex2f( x1+x, y1+y );
		glVertex2f( x2+x, y2+y );
		glEnd();
	}
}

// 円の描画（塗りつぶし）
template <typename T> void Circle2DFill(T r,int x,int y)
{
	for (T th1 = 0.0;  th1 <= 360.0;  th1 = th1 + 1.0)
	{
		T th2 = th1 + 10.0;
		T th1_rad = th1 / 180.0 * M_PI;
		T th2_rad = th2 / 180.0 * M_PI;

		T x1 = r * cos(th1_rad);
		T y1 = r * sin(th1_rad);
		T x2 = r * cos(th2_rad);
		T y2 = r * sin(th2_rad);

		glBegin(GL_TRIANGLES);
		glVertex2f( x, y );
		glVertex2f( x1+x, y1+y );
		glVertex2f( x2+x, y2+y );
		glEnd();
	}
}

template <typename T> void SetGamma( std::vector<T> &Gam )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T max = (T)(0.0);
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i != j)
			{
				thrust::complex<T> vf, dfi, dfj;

				Horner( Cef, Zrs[i], vf, dfi );
				Horner( Cef, Zrs[j], vf, dfj );

				if (max < abs(dfi / dfj))
				{
					max = abs(dfi / dfj);
				}
			}
		}
		Gam[i] = max;
	}
}

template <typename T> T fAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	return ((T)(Zrs.size()) -1.0) * Gamma * pow(alp,(T)(p-1)) - (1.0-alp)/(1.0+3.0*alp);
}

template <typename T> T dAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	T tmp = ((T)(Zrs.size()) -1.0) * Gamma * ((T)(p) -1.0) * pow(alp,(T)(p-2));
	return  tmp + 4.0/(1.0+6.0*alp+9.0*alp*alp);
}

template <typename T> void GetAlpha( const std::vector<T> Gam, std::vector<T> &Alp )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T alp = (T)(1.0);
		int count = 0;

		while ((count < MAXIT) && (abs(fAlp(P,Gam[i],alp)) > EPS))
		{
			alp -= fAlp(P,Gam[i],alp) / dAlp(P,Gam[i],alp);
			count++;
		}
		if (count == MAXIT)
		{
			std::cerr << "No convergence in GetAlpha\n";
			std::exit (EXIT_FAILURE);
		}
		Alp[i] = alp;
	}
}

// Apollonius円の描画
template <typename T> void DrawApollonius( const int i, const int j, const T alp )
{
	assert(alp<1.0);

	thrust::complex<T> center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
	double radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);

	Circle2D( radius, (int)(center.real()), (int)(center.imag()) );
}

// Apollonius領域の描画
template <typename T> void DrawApRegion( const T alp )
{
	const int pts = 20;    // 円周上の点数
	thrust::complex<T> center;
	T radius;
	T start, end, tic;

	int i, j;

	// p=2
	{
		// z_0, z_1
		i = 0; j=1;
		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
		start = -1.41075;
		tic = -2.0*start / pts;

		glBegin(GL_LINE_STRIP);
		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
		for (int t=1; t<=pts; t++)
		{
			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
		}
		glEnd();

		// z_0, z_2
		i = 0; j=2;
		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
		start = 1.73084;
		tic = 2.0*(M_PI - start) / pts;

		glBegin(GL_LINE_STRIP);
		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
		for (int t=1; t<=pts; t++)
		{
			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
		}
		glEnd();
	}

	// p=4
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.22945;
//		tic = -2.0*start / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 1.91214;
//		tic = 2.0*(M_PI - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=8
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.08417;
//		tic = -2.0*start / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 2.05743;
//		tic = 2.0*(M_PI - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=16
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -0.0309316;
//		end   = 0.976624;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start =  2.16497;
//		end   =  2*M_PI - 3.11066;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_3
//		i = 0; j=3;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -1.0758;
//		end   = -0.928056;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_4
//		i = 0; j=4;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -2.21354;
//		end   = -2.06579;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}

	// p=32
//	{
//		// z_0, z_1
//		i = 0; j=1;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = 0.325413;
//		end   = 0.902587;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_2
//		i = 0; j=2;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start =  2.23901;
//		end   =  2.81618;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_3
//		i = 0; j=3;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -0.903981;
//		end   = -1.01722;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//
//		// z_0, z_4
//		i = 0; j=4;
//		center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
//		radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
//		start = -2.12437;
//		end   = -2.23761;
//		tic = (end - start) / pts;
//
//		glBegin(GL_LINE_STRIP);
//		glVertex2d( center.real() + radius*cos(start), center.imag() + radius*sin(start) );
//		for (int t=1; t<=pts; t++)
//		{
//			glVertex2d( center.real() + radius*cos( start+tic*t )  , center.imag() + radius*sin( start+tic*t ) );
//		}
//		glEnd();
//	}
}

template <typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)(MAXIT);

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs( z - *itr) < min)
		{
			min = abs( z - *itr);
			col = i;
		}
		i++;
	}
	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double min = (double)(MAXIT);
	double max = 0.0;

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(Zrs,Cef,P,z0,count,er);

			double bright;
			if (count > MAXIT)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				//bright = double(MAXIT - (count-1)) / double(MAXIT);
				bright = double(MAXIT - (count)) / double(MAXIT);
			}
//			std::cout << bright << std::endl;
			if (bright > max)
				max = bright;
			if (bright < min)
				min = bright;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
//	std::cout << "min = " << min << ", max = " << max << std::endl;
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	glColor3d(1.0,1.0,1.0);   // 白の点を描画
	glLineWidth(1.0);
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
		Circle2DFill( (double)(0.05), (*itr).real(), (*itr).imag() );
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apolloniusの描画
	SetGamma( Gam );       // Γ
	GetAlpha( Gam, Alp );  // α

	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(1.0);         // 線の太さ（ディフォルトは1.0）
	for (int i=0; i<Zrs.size(); i++)
	{
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i!=j)
				DrawApollonius(i,j,Alp[i]);
		}
	}
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apollonius領域の描画
	SetGamma( Gam );       // Γ
	GetAlpha( Gam, Alp );  // α

	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(2.0);         // 線の太さ（縮小時は4.0にする）
	DrawApRegion( Alp[0] );
	//////////////////////////////////////////////

	glFlush();
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	if (argc<2)
	{
		std::cerr << "Usage: a.out [Order]\n";
		exit (EXIT_FAILURE);
	}
	P = atoi(argv[1]);  // Nourein法の次数

	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1000,1000);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
