#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <GLUT/glut.h>
#include <thrust/complex.h>
#include <thrust/host_vector.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
#define RMAX 2000     // 複素平面の分割数
#define ORD  32        // Nourein法の次数

#define NFP 5 // 零点の数
thrust::complex<double> fps[NFP];

void setZero( thrust::complex<double> *fps )
{
	fps[0] = thrust::complex<double> (  0.0,  1.0 );
	fps[1] = thrust::complex<double> (  1.0,  2.0 );
	fps[2] = thrust::complex<double> ( -1.0,  2.0 );
	fps[3] = thrust::complex<double> (  3.0, -3.0 );
	fps[4] = thrust::complex<double> ( -3.0, -3.0 );
}

// Polynomial
thrust::complex<double> vf( thrust::complex<double> z )
{
	thrust::complex<double> iu = thrust::complex<double> ( 0.0, 1.0 );
	return z*z*z*z*z + iu*z*z*z*z + + 3.0*z*z*z + 41.0*iu*z*z + 132.0*z -90.0*iu;
}

// derived function of the polynomial
thrust::complex<double> df( thrust::complex<double> z )
{
	thrust::complex<double> iu = thrust::complex<double> ( 0.0, 1.0 );
	return 5.0*z*z*z*z + 4.0*iu*z*z*z + 9.0*z*z + 82.0*iu*z + 132.0;
}

// Nourein subfunction
thrust::complex<double> vc( const int K, thrust::complex<double> z )
{
	thrust::complex<double> f = thrust::complex<double> (0.0,0.0);;

	for (int i=0; i<NFP; i++)
	{
		thrust::complex<double> tmp = thrust::complex<double> (1.0,0.0);

		// tmp = (z_i -z)^{k+1}
		for (int k=0; k<=K; k++)
		{
			tmp = tmp * (fps[i] - z);
		}
		// tmp = -1.0 /  (z_i -z)^{k+1}
		tmp = -1.0 / tmp;

		f += ( 1.0 / df(fps[i]) )*tmp;
	}
	return f;
}

thrust::complex<double> Nourein( const int p, thrust::complex<double> z, int &count, double &er )
{
	assert(p>=2);

	count = 0;

	while ((count < MAXIT) && (abs(vf(z)) > EPS))
	{
		z += vc(p-2,z) / vc(p-1,z);
		count++;
	}
	er = abs(vf(z));

	return z;
}

void DrawApollonius( int i, int j, double alp )
{
	const int pts = 180;    // 円周上の点数

	thrust::complex<double> center = (fps[i] - alp*alp*fps[j]) / (1.0 - alp*alp);
	double radius = alp*abs(fps[i] - fps[j]) / (1.0 - alp*alp);
	double tic = (double)(2.0*M_PI / pts);

	// Z_i の描画
	glColor3d(1.0,1.0,1.0);   // 白の点を描画
	glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
	glBegin(GL_POINTS);
	glVertex2d( fps[i].real(), fps[i].imag() );
	glEnd();

	// Apollonius円の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(2.0);         // 線の太さ（ディフォルトは1.0）

	glBegin(GL_LINE_LOOP);
	for (int i=1; i<pts; i++)
	{
		glVertex2d( center.real() + radius*cos( tic*i )  , center.imag() + radius*sin( tic*i ) );
	}
	glEnd();
	glFlush();
}

int FixPoint( thrust::complex<double> z )
{
	int col = 0;
	double min = (double)MAXIT;

	for (int i=0; i<NFP; i++)
	{
		if (abs(z - fps[i]) < min)
		{
			min = abs(z - fps[i]);
			col = i;
		}
	}

	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(1.0, 1.0, 1.0, 1.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(ORD,z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}
			// 明るさの補正
//			bright += 0.2;

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	for (int i=0; i<NFP; i++)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( fps[i].real(), fps[i].imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	glFlush();

}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	setZero(fps);     // 零点のセット

	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
