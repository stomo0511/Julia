#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
//#define RMAX 1000     // 複素平面の分割数（ iMacでは 2000 とする）
#define RMAX 500     // 複素平面の分割数（ iMacでは 2000 とする）

int P;  // Nourein法の次数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  0.0,  1.0 ),
	thrust::complex<double> (  1.0,  2.0 ),
	thrust::complex<double> ( -1.0,  2.0 ),
	thrust::complex<double> (  3.0, -3.0 ),
	thrust::complex<double> ( -3.0, -3.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0,   0.0 ),  // z^5
	thrust::complex<double> (  0.0,   1.0 ),  // Z^4
	thrust::complex<double> (  3.0,   0.0 ),  // Z^3
	thrust::complex<double> (  0.0,  41.0 ),  // z^2
	thrust::complex<double> (132.0,   0.0 ),  // z^1
	thrust::complex<double> (  0.0, -90.0 )   // z^0
};

std::vector<double> Gam( Zrs.size() );  // Γ
std::vector<double> Alp( Zrs.size() );  // α

// Hornet method for polynomial
template<typename T> void Horner( std::vector< thrust::complex<T> > cf, thrust::complex<T> z,
					thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = Cef[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = Cef.begin()+1; itr < Cef.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

// Nourein subfunction
template <typename T> thrust::complex<T> vc( const int K, thrust::complex<T> z )
{
	thrust::complex<T> tmp = thrust::complex<T> (0.0,0.0);;

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		thrust::complex<T> vf, df;
		Horner( Cef, *itr, vf, df );

		// tmp *= (1/f'(z_i) (-1 / (z_i -z)^{K+1})
		tmp += ( (T)(1.0) / df )*( (T)(-1.0) / pow( (*itr - z), (T)(K+1) ));
	}
	return tmp;
}

template <typename T> thrust::complex<T> Nourein( const int p, thrust::complex<T> z, int &count, T &er )
{
	assert(p>=2);

	thrust::complex<T> vf, df;
	Horner( Cef, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z += vc(p-2,z) / vc(p-1,z);
		Horner( Cef, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

template <typename T> void SetGamma( std::vector<T> &Gam )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T max = (T)(0.0);
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i != j)
			{
				thrust::complex<T> vf, dfi, dfj;

				Horner( Cef, Zrs[i], vf, dfi );
				Horner( Cef, Zrs[j], vf, dfj );

				if (max < abs(dfi / dfj))
				{
					max = abs(dfi / dfj);
				}
			}
		}
		Gam[i] = max;
	}
}

template <typename T> T fAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	return ((T)(Zrs.size()) -1.0) * Gamma * pow(alp,(T)(p-1)) - (1.0-alp)/(1.0+3.0*alp);
}

template <typename T> T dAlp( const int p, const T Gamma, T alp )
{
	assert(p>=2);

	T tmp = ((T)(Zrs.size()) -1.0) * Gamma * ((T)(p) -1.0) * pow(alp,(T)(p-2));
	return  tmp + 4.0/(1.0+6.0*alp+9.0*alp*alp);
}

template <typename T> void GetAlpha( const std::vector<T> Gam, std::vector<T> &Alp )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T alp = (T)(1.0);
		int count = 0;

		while ((count < MAXIT) && (abs(fAlp(P,Gam[i],alp)) > EPS))
		{
			alp -= fAlp(P,Gam[i],alp) / dAlp(P,Gam[i],alp);
			count++;
		}
		if (count == MAXIT)
		{
			std::cerr << "No convergence in GetAlpha\n";
			std::exit (EXIT_FAILURE);
		}
		Alp[i] = alp;
	}
}

template <typename T> void DrawApollonius( const int i, const int j, const T alp )
{
	const int pts = 180;    // 円周上の点数

	thrust::complex<T> center = (Zrs[i] - alp*alp*Zrs[j]) / (1.0 - alp*alp);
	double radius = alp*abs(Zrs[i] - Zrs[j]) / (1.0 - alp*alp);
	double tic = (double)(2.0*M_PI / pts);

	// Apollonius円の描画
	glColor3d(1.0,1.0,1.0);   // 白の円を描画
	glLineWidth(1.0);         // 線の太さ（ディフォルトは1.0）

	glBegin(GL_LINE_LOOP);
	for (int i=1; i<pts; i++)
	{
		glVertex2d( center.real() + radius*cos( tic*i )  , center.imag() + radius*sin( tic*i ) );
	}
	glEnd();
	glFlush();
}

template <typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)(MAXIT);

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs( z - *itr) < min)
		{
			min = abs( z - *itr);
			col = i;
		}
		i++;
	}
	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(0.0, 0.0, 0.0, 0.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(P,z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( (*itr).real(), (*itr).imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// Apolloniusの描画
	SetGamma( Gam );       // Γ
	GetAlpha( Gam, Alp );  // α
//	for (int i=0; i<Zrs.size(); i++)
	int i=0;
	{
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i!=j)
				DrawApollonius(i,j,Alp[i]);
		}
	}
	//////////////////////////////////////////////

	glFlush();
}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	if (argc<2)
	{
		std::cerr << "Usage: a.out [Order]\n";
		exit (EXIT_FAILURE);
	}
	P = atoi(argv[1]);  // Nourein法の次数

	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
