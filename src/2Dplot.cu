#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 2Dplot.cu
 Author      : stomo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>
#include <algorithm>
#include <vector>
#include <GLUT/glut.h>
#include <thrust/complex.h>

#define EPS 0.000001  // 停止判定
#define MAXIT 30      // 最大反復回数
#define ZMAX 4.0      // 初期値の最大絶対値
#define ZOOM 200      // 拡大率
//#define RMAX 1000     // 複素平面の分割数（ iMacでは 2000 とする）
#define RMAX 500     // 複素平面の分割数（ iMacでは 2000 とする）
#define ORD  2        // Nourein法の次数

// Zeros
std::vector< thrust::complex<double> > Zrs {
	thrust::complex<double> (  0.0,  1.0 ),
	thrust::complex<double> (  1.0,  2.0 ),
	thrust::complex<double> ( -1.0,  2.0 ),
	thrust::complex<double> (  3.0, -3.0 ),
	thrust::complex<double> ( -3.0, -3.0 )
};

// Coefficients
std::vector< thrust::complex<double> > Cef {
	thrust::complex<double> (  1.0,   0.0 ),  // z^5
	thrust::complex<double> (  0.0,   1.0 ),  // Z^4
	thrust::complex<double> (  3.0,   0.0 ),  // Z^3
	thrust::complex<double> (  0.0,  41.0 ),  // z^2
	thrust::complex<double> (132.0,   0.0 ),  // z^1
	thrust::complex<double> (  0.0, -90.0 )   // z^0
};

// Hornet method for polynomial
template<typename T> void Horner( std::vector< thrust::complex<T> > cf, thrust::complex<T> z,
					thrust::complex<T> &vf, thrust::complex<T> &df )
{
	vf = Cef[0];
	df = thrust::complex<T> (0.0,0.0);
	thrust::complex<T> tmp;

    for(auto itr = Cef.begin()+1; itr < Cef.end(); ++itr)
    {
    	tmp = vf;
    	vf = vf*z + *itr;
    	df = df*z + tmp;
    }
}

// Nourein subfunction
template <typename T> thrust::complex<T> vc( const int K, thrust::complex<T> z )
{
	thrust::complex<T> tmp = thrust::complex<T> (0.0,0.0);;

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		thrust::complex<T> vf, df;
		Horner( Cef, *itr, vf, df );

		// tmp *= (1/f'(z_i) (-1 / (z_i -z)^{K+1})
		tmp += ( (T)(1.0) / df )*( (T)(-1.0) / pow( (*itr - z), (T)(K+1) ));
	}
	return tmp;
}

template <typename T> thrust::complex<T> Nourein( const int p, thrust::complex<T> z, int &count, T &er )
{
	assert(p>=2);

	thrust::complex<T> vf, df;
	Horner( Cef, z, vf, df );
	count = 0;

	while ((count < MAXIT) && (abs(vf) > EPS))
	{
		z += vc(p-2,z) / vc(p-1,z);
		Horner( Cef, z, vf, df );
		count++;
	}
	er = abs(vf);

	return z;
}

template <typename T> void SetGamma( std::vector<T> &Gam )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		T max = (T)(0.0);
		for (int j=0; j<Zrs.size(); j++)
		{
			if (i != j)
			{
				thrust::complex<T> vf, dfi, dfj;

				Horner( Cef, Zrs[i], vf, dfi );
				Horner( Cef, Zrs[j], vf, dfj );

				if (max < abs(dfi / dfj))
				{
					max = abs(dfi / dfj);
				}
			}
		}
		Gam[i] = max;
	}
}

template <typename T> void GetAlpha( std::vector<T> Gam, std::vector<T> &Alp )
{
	for (int i=0; i<Zrs.size(); i++)
	{
		(Zrs.size() -1.0);
	}
}
//void DrawApollonius( int i, int j, double alp )
//{
//	const int pts = 180;    // 円周上の点数
//
//	thrust::complex<T> center = (fps[i] - alp*alp*fps[j]) / (1.0 - alp*alp);
//	double radius = alp*abs(fps[i] - fps[j]) / (1.0 - alp*alp);
//	double tic = (double)(2.0*M_PI / pts);
//
//	// Z_i の描画
//	glColor3d(1.0,1.0,1.0);   // 白の点を描画
//	glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
//	glBegin(GL_POINTS);
//	glVertex2d( fps[i].real(), fps[i].imag() );
//	glEnd();
//
//	// Apollonius円の描画
//	glColor3d(1.0,1.0,1.0);   // 白の円を描画
//	glLineWidth(2.0);         // 線の太さ（ディフォルトは1.0）
//
//	glBegin(GL_LINE_LOOP);
//	for (int i=1; i<pts; i++)
//	{
//		glVertex2d( center.real() + radius*cos( tic*i )  , center.imag() + radius*sin( tic*i ) );
//	}
//	glEnd();
//	glFlush();
//}

template <typename T> int FixPoint( thrust::complex<T> z )
{
	int i = 0;
	int col = 0;
	double min = (double)(MAXIT);

	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr )
	{
		if (abs( z - *itr) < min)
		{
			min = abs( z - *itr);
			col = i;
		}
		i++;
	}
	return col;
}

void display(void)
{
	//////////////////////////////////////////////
	// 背景を白に
	glClearColor(0.0, 0.0, 0.0, 0.0); // 塗りつぶしの色を指定
	glClear(GL_COLOR_BUFFER_BIT);     // 塗りつぶし

	//////////////////////////////////////////////
	// 点の描画
	glBegin(GL_POINTS);

	double x = (double)(-ZMAX);
	for (int i=0; i<RMAX; i++)
	{
		double y = (double)(-ZMAX);
		for (int j=0; j<RMAX; j++)
		{
			int count;
			double er;
			thrust::complex<double> z0 = thrust::complex<double>( x, y );
			thrust::complex<double> z = Nourein(ORD,z0,count,er);

			int grad = 16;  // 明るさの階調
			double bright;
			if (count > grad)
				bright = 0.0;
			else
			{
				// 反復回数1回が最も明るく（bright=1）となるように修正（count-1）
				bright = double(grad - (count-1)) / double(grad);
			}

			switch( FixPoint(z) )  // 塗りつぶし色の設定
			{
			case 0:
				glColor3d(bright,0.0,0.0);
				break;
			case 1:
				glColor3d(0.0,bright,0.0);
				break;
			case 2:
				glColor3d(0.0,0.0,bright);
				break;
			case 3:
				glColor3d(bright,0.0,bright);
				break;
			case 4:
				glColor3d(0.0,bright,bright);
				break;
			default:
				glColor3d(0.0,0.0,0.0);
				break;
			}

			glVertex2d( z0.real(), z0.imag() );  // 点の描画
			y += (double)(2*ZMAX / RMAX);
		}
		x += (double)(2*ZMAX / RMAX);
	}
	glEnd();
	//////////////////////////////////////////////

	//////////////////////////////////////////////
	// 零点の描画
	for (auto itr = Zrs.begin(); itr < Zrs.end(); ++itr)
	{
		glColor3d(1.0,1.0,1.0);   // 白の点を描画
		glPointSize(8.0);      // 点の大きさ（ディフォルトは1.0)
		glBegin(GL_POINTS);
		glVertex2d( (*itr).real(), (*itr).imag() );
		glEnd();
	}
	//////////////////////////////////////////////

	glFlush();

}

void resize(int w, int h)
{
	// Window全体をView portにする
	glViewport(0,0,w,h);

	// 変換行列の初期化
	glLoadIdentity();

	// Screen上の表示領域をView portの大きさに比例させる
	glOrtho( -(double)w/ZOOM, (double)w/ZOOM, -(double)h/ZOOM, (double)h/ZOOM, -1.0, 1.0);
}

int main(int argc, char *argv[])
{
	std::vector<double> Gam( Zrs.size() );
	SetGamma( Gam );

    for(auto itr = Gam.begin(); itr < Gam.end(); ++itr)
    {
    	std::cout << *itr << std::endl;
    }

	glutInit(&argc, argv);          // OpenGL初期化
	glutInitWindowSize(1100,1100);  // 初期Windowサイズ指定
	glutCreateWindow(argv[0]);      // Windowを開く
	glutDisplayFunc(display);       // Windowに描画
	glutReshapeFunc(resize);
	glutMainLoop();                 // イベント待ち

	return EXIT_SUCCESS;
}
